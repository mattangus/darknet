#include "hip/hip_runtime.h"
#include "gpu/dark.hpp"
#include <stdexcept>
#include <iostream>

namespace darknet
{
namespace gpu
{
    dim3 cuda_gridsize(size_t n, size_t block_size){
        size_t k = (n-1) / block_size + 1;
        size_t x = k;
        size_t y = 1;
        if(x > 65535){
            x = ceil(sqrt(k));
            y = (n-1)/(x*block_size) + 1;
        }
        
        dim3 d;
        d.x = x;
        d.y = y;
        d.z = 1;

        return d;
    }

    size_t get_number_of_blocks(size_t array_size, size_t block_size)
    {
        return array_size / block_size + ((array_size % block_size > 0) ? 1 : 0);
    }


    void check_error_extended(hipError_t status, const std::string file, int line, const std::string date_time)
    {
        if (status != hipSuccess) {
            std::cerr << "CUDA status Error: file: " << file << " : line: " << line << " : build time: " << date_time << std::endl;
            throw std::runtime_error(hipGetErrorString(status));
        }
    }

// #ifdef CUDNN
    void cudnn_check_error_extended(hipdnnStatus_t status, const std::string file, int line, const std::string date_time)
    {
        if (status != HIPDNN_STATUS_SUCCESS) {
            std::cerr << "cuDNN status Error: file: " << file << " : line: " << line << " : build time: " << date_time << std::endl;
            throw std::runtime_error(hipdnnGetErrorString(status));
        }
    }
// #endif
} // namespace gpu
} // namespace darknet
